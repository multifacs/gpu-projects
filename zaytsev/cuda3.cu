#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <omp.h>

__global__ void float_matrix_multiplication_kernel(const int m, const int n, const int k, float* x, float* y, float* z)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < k && row < m) {
        for (int i = 0; i < n; i++)
            z[row * k + col] += x[row * n + i] * y[i * k + col];
    }
}

__global__ void block_float_matrix_multiplication_kernel(const int m, const int n, const int k, float* x, float* y, float* z)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    float res = 0;
    __shared__ float a_block[16 * 16];
    __shared__ float b_block[16 * 16];
    if (col < k && row < m) {
        for (int i = 0; i < n; i += blockDim.y)
        {
            a_block[threadIdx.y * blockDim.x + threadIdx.x] = x[(blockIdx.y * blockDim.y + threadIdx.y) * n + (i + threadIdx.x)];
            b_block[threadIdx.y * blockDim.y + threadIdx.x] = y[(i + threadIdx.y) * k + (blockIdx.x * blockDim.x + threadIdx.x)];
            __syncthreads();
            for (int j = 0; j < blockDim.x; j++) {
                res += a_block[threadIdx.y * blockDim.y + j] * b_block[j * blockDim.y + threadIdx.x];
            }
            __syncthreads();
        }
        z[row * k + col] += res;
    }
}

void float_matrix_multiplication_cuda(const int m, const int n, const int k, const float* x, const float* y, float* z, const dim3 dimGrid, const dim3 dimBlock)
{

    hipError_t cudaStatus;

    float* gpuX, * gpuY, * gpuZ;
    cudaStatus = hipMalloc((void**)&gpuX, n * m * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(gpuX) faild\n");
        return;
    }
    cudaStatus = hipMalloc((void**)&gpuY, n * k * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(gpuY) faild\n");
        return;
    }
    cudaStatus = hipMalloc((void**)&gpuZ, m * k * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(gpuZ) faild\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuX, x, n * m * sizeof(float),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy(gpuX) faild\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuY, y, n * k * sizeof(float),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy(gpuY) faild\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuZ, z, m * k * sizeof(float),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy(gpuZ) faild\n");
        return;
    }

    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    float_matrix_multiplication_kernel << <dimGrid, dimBlock >> > (m, n, k, gpuX, gpuY, gpuZ);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("GPU = %f", gpuTime / 1000);
    printf("\n");

    cudaStatus = hipMemcpy(z, gpuZ, m * k * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy(gpuZ) faild\n");
        return;
    }

    hipFree(gpuX); hipFree(gpuY); hipFree(gpuZ);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return;

}

void block_float_matrix_multiplication_cuda(const int m, const int n, const int k, const float* x, const float* y, float* z, const dim3 dimGrid, const dim3 dimBlock)
{

    hipError_t cudaStatus;

    float* gpuX, * gpuY, * gpuZ;
    cudaStatus = hipMalloc((void**)&gpuX, n * m * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(gpuX) faild\n");
        return;
    }
    cudaStatus = hipMalloc((void**)&gpuY, n * k * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(gpuY) faild\n");
        return;
    }
    cudaStatus = hipMalloc((void**)&gpuZ, m * k * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc(gpuZ) faild\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuX, x, n * m * sizeof(float),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy(gpuX) faild\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuY, y, n * k * sizeof(float),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy(gpuY) faild\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuZ, z, m * k * sizeof(float),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy(gpuZ) faild\n");
        return;
    }

    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    block_float_matrix_multiplication_kernel << <dimGrid, dimBlock >> > (m, n, k, gpuX, gpuY, gpuZ);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("GEMM GPU time = %f", gpuTime / 1000);
    printf("\n");

    cudaStatus = hipMemcpy(z, gpuZ, m * k * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy(gpuZ) faild\n");
        return;
    }

    hipFree(gpuX); hipFree(gpuY); hipFree(gpuZ);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return;

}

void float_matrix_multiplication(const int m, const int n, const int k, float* x, float* y, float* z)
{
    for (int i = 0; i < m; ++i)
        for (int p = 0; p < k; ++p)
            for (int j = 0; j < n; ++j)
                z[i * k + p] += x[i * n + j] * y[j * k + p];
}

void float_matrix_multiplication_omp(const int m, const int n, const int k, float* x, float* y, float* z)
{
#pragma omp parallel for
    for (int i = 0; i < m; ++i)
    {
        float* c = z + i * n;
        for (int j = 0; j < n; ++j)
            c[j] = 0;
        for (int p = 0; p < k; ++p)
        {
            const float* b = y + p * n;
            float a = x[i * k + p];
            for (int j = 0; j < n; ++j)
                c[j] += a * b[j];
        }
    }
}

int main()
{
    const int n_1 = 512;
    const int m_1 = 512;
    const int k_1 = 512;

    float* x_1 = new float[n_1 * m_1];
    float* y_1 = new float[n_1 * k_1];
    float* z_1 = new float[m_1 * k_1];

    for (int i = 0; i < n_1 * m_1; i++)
    {
        x_1[i] = 1.0;
    }
    for (int i = 0; i < n_1 * k_1; i++)
    {
        y_1[i] = 1.0;
    }
    for (int i = 0; i < m_1 * k_1; i++)
    {
        z_1[i] = 0.0;
    }

    float start_1 = omp_get_wtime();

    float_matrix_multiplication(m_1, n_1, k_1, x_1, y_1, z_1);

    float end_1 = omp_get_wtime();

    printf("Seq. = %f", end_1 - start_1);
    printf("\n");

    delete[] x_1, delete[] y_1, delete[] z_1;

    x_1 = new float[n_1 * m_1];
    y_1 = new float[n_1 * k_1];
    z_1 = new float[m_1 * k_1];

    for (int i = 0; i < n_1 * m_1; i++)
    {
        x_1[i] = 1.0;
    }
    for (int i = 0; i < n_1 * k_1; i++)
    {
        y_1[i] = 1.0;
    }
    for (int i = 0; i < m_1 * k_1; i++)
    {
        z_1[i] = 0.0;
    }

    start_1 = omp_get_wtime();

    float_matrix_multiplication_omp(m_1, n_1, k_1, x_1, y_1, z_1);

    end_1 = omp_get_wtime();

    printf("OMP = %f", end_1 - start_1);
    printf("\n");

    delete[] x_1, delete[] y_1, delete[] z_1;

    const int n_0f = 512;
    const int m_0f = 512;
    const int k_0f = 512;

    float* x_0f = new float[n_0f * m_0f];
    float* y_0f = new float[n_0f * k_0f];
    float* z_0f = new float[m_0f * k_0f];

    dim3 dimBlockf(16, 16);
    dim3 dimGridf((m_0f + dimBlockf.x - 1) / dimBlockf.x, (k_0f + dimBlockf.y - 1) / dimBlockf.y);

    for (int i = 0; i < n_0f * m_0f; i++)
    {
        x_0f[i] = 1.0;
    }
    for (int i = 0; i < n_0f * k_0f; i++)
    {
        y_0f[i] = 1.0;
    }
    for (int i = 0; i < m_0f * k_0f; i++)
    {
        z_0f[i] = 0.0;
    }

    float_matrix_multiplication_cuda(m_0f, n_0f, k_0f, x_0f, y_0f, z_0f, dimGridf, dimBlockf);

    delete[] x_0f, delete[] y_0f, delete[] z_0f;


    const int n_1f = 512;
    const int m_1f = 512;
    const int k_1f = 512;

    dim3 dimBlock_1f(16, 16);
    dim3 dimGrid_1f((m_1f + dimBlock_1f.x - 1) / dimBlock_1f.x, (k_1f + dimBlock_1f.y - 1) / dimBlock_1f.y);

    float* x_1f = new float[n_1f * m_1f];
    float* y_1f = new float[n_1f * k_1f];
    float* z_1f = new float[m_1f * k_1f];

    for (int i = 0; i < n_1f * m_1f; i++)
    {
        x_1f[i] = 1.0;
    }
    for (int i = 0; i < n_1f * k_1f; i++)
    {
        y_1f[i] = 1.0;
    }
    for (int i = 0; i < m_1f * k_1f; i++)
    {
        z_1f[i] = 0.0;
    }

    block_float_matrix_multiplication_cuda(m_1f, n_1f, k_1f, x_1f, y_1f, z_1f, dimGrid_1f, dimBlock_1f);

    delete[] x_1f, delete[] y_1f, delete[] z_1f;

    return 0;
}
