
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime_api.h>
#include <omp.h>

__global__ void saxpy_kernel(const int n, const float a, float *x, const int incx, float *y, const int incy)
{
    const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
    const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        y[biasy + i * incy] += a * x[biasx + i * incx];
    }
}

__global__ void daxpy_kernel(const int n, const double a, double *x, const int incx, double *y, const int incy)
{
    const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
    const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        y[biasy + i * incy] += a * x[biasx + i * incx];
    }
}

void saxpy_gpu(const int n, const float a, float *x, const int incx, float *y, const int incy, const int numBlocks, const int blocksSize)
{
    hipError_t cudaStatus;
    int sizeX = 1 + (n - 1) * abs(incx);
    int sizeY = 1 + (n - 1) * abs(incy);

    float *gpuX;
    cudaStatus = hipMalloc((void **)&gpuX, sizeX * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMalloc(gpuX) faild f1\n");
        return;
    }

    float *gpuY;
    cudaStatus = hipMalloc((void **)&gpuY, sizeY * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMalloc(gpuY) faild f2\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuX, x, sizeX * sizeof(float),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMemcpy(gpuX) faild\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuY, y, sizeY * sizeof(float),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMemcpy(gpuY) faild\n");
        return;
    }

    hipEvent_t startF, stopF;
    float gpuTimeF = 0.0f;

    hipEventCreate(&startF);
    hipEventCreate(&stopF);
    hipEventRecord(startF, 0);

    saxpy_kernel<<<numBlocks, blocksSize>>>(n, a, gpuX, incx, gpuY, incy);

    hipEventRecord(stopF, 0);
    hipEventSynchronize(stopF);
    hipEventElapsedTime(&gpuTimeF, startF, stopF);

    printf("GPU float time = %f", gpuTimeF / 1000);

    cudaStatus = hipMemcpy(y, gpuY, sizeY * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMemcpy(gpuY) faild\n");
        return;
    }

    hipFree(gpuX);
    hipFree(gpuY);
    hipEventDestroy(startF);
    hipEventDestroy(stopF);
    return;
}

void daxpy_gpu(const int n, const double a, double *x, const int incx, double *y, const int incy, const int numBlocks, const int blocksSize)
{
    hipError_t cudaStatus;
    int sizeX = 1 + (n - 1) * abs(incx);
    int sizeY = 1 + (n - 1) * abs(incy);

    double *gpuX;
    cudaStatus = hipMalloc((void **)&gpuX, sizeX * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMalloc(gpuX) faild d1\n");
        return;
    }

    double *gpuY;
    cudaStatus = hipMalloc((void **)&gpuY, sizeY * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMalloc(gpuY) faild d2\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuX, x, sizeX * sizeof(double),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMemcpy(gpuX) faild\n");
        return;
    }

    cudaStatus = hipMemcpy(gpuY, y, sizeY * sizeof(double),
                            hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMemcpy(gpuY) faild\n");
        return;
    }

    hipEvent_t startD, stopD;
    float gpuTimeD = 0.0f;

    hipEventCreate(&startD);
    hipEventCreate(&stopD);
    hipEventRecord(startD, 0);

    daxpy_kernel<<<numBlocks, blocksSize>>>(n, a, gpuX, incx, gpuY, incy);

    hipEventRecord(stopD, 0);
    hipEventSynchronize(stopD);
    hipEventElapsedTime(&gpuTimeD, startD, stopD);

    printf("GPU double time = %f", gpuTimeD / 1000);

    cudaStatus = hipMemcpy(y, gpuY, sizeY * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "cudaMemcpy(gpuY) faild\n");
        return;
    }

    hipFree(gpuX);
    hipFree(gpuY);
    hipEventDestroy(startD);
    hipEventDestroy(stopD);
    return;
}

template <typename t>
bool comp(t *a1, t *a2, size_t size)
{
    for (size_t i = 0; i < size; i++)
    {
        if (a1[i] != a2[i])
            return false;
    }
    return true;
}

void saxpy(const int n, const float a, float *x, const int incx, float *y, const int incy)
{
    const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
    const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

    for (size_t i = 0; i < n; i++)
    {
        y[biasy + i * incy] += a * x[biasx + i * incx];
    }
}

void daxpy(const int n, const double a, double *x, const int incx, double *y, const int incy)
{
    const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
    const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

    for (size_t i = 0; i < n; i++)
    {
        y[biasy + i * incy] += a * x[biasx + i * incx];
    }
}

void saxpy_omp(const int n, const float a, float *x, const int incx, float *y, const int incy)
{
    const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
    const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

#pragma omp parallel for num_threads(4)
    for (int i = 0; i < n; i++)
    {
        y[biasy + i * incy] += a * x[biasx + i * incx];
    }
}

void daxpy_omp(const int n, const double a, double *x, const int incx, double *y, const int incy)
{
    const int biasx = incx < 0 ? (n - 1) * abs(incx) : 0;
    const int biasy = incy < 0 ? (n - 1) * abs(incy) : 0;

#pragma omp parallel for num_threads(4)
    for (int i = 0; i < n; i++)
    {
        y[biasy + i * incy] += a * x[biasx + i * incx];
    }
}

int main()
{
    const int n = 5000000; // 1e7;
    const int incx = 10;
    const int incy = 10;
    const int sizeX = 1 + (n - 1) * abs(incx);
    const int sizeY = 1 + (n - 1) * abs(incy);
		int block_size;
    int num_blocks;

    printf("n = %d\n", n);
		printf("\n");


    const float aF = 10.0f;

    float *xF = new float[sizeX];
    float *yF = new float[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xF[i] = 5.0f;
        yF[i] = 1.0f;
    }

    double startF = omp_get_wtime();

    saxpy(n, aF, xF, incx, yF, incy);

    double endF = omp_get_wtime();

    printf("Seq. float = %f", endF - startF);
    printf("\n");

    delete[] xF;
    delete[] yF;

    const double aD = 10.0;

    double *xD = new double[sizeX];
    double *yD = new double[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xD[i] = 5.0;
        yD[i] = 1.0;
    }

    double startD = omp_get_wtime();

    daxpy(n, aD, xD, incx, yD, incy);

    double endD = omp_get_wtime();

    printf("Seq. double = %f", endD - startD);
    printf("\n");
    printf("\n");

    delete[] xD;
    delete[] yD;

    const float aPF = 10.0f;

    float *xPF = new float[sizeX];
    float *yPF = new float[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xPF[i] = 5.0f;
        yPF[i] = 1.0f;
    }

    double startPF = omp_get_wtime();

    saxpy_omp(n, aPF, xPF, incx, yPF, incy);

    double endPF = omp_get_wtime();

    printf("Parallel float = %f", endPF - startPF);
    printf("\n");

    delete[] xPF;
    delete[] yPF;

    const double aPD = 10.0;

    double *xPD = new double[sizeX];
    double *yPD = new double[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xPD[i] = 5.0;
        yPD[i] = 1.0;
    }

    double startPD = omp_get_wtime();

    daxpy_omp(n, aPD, xPD, incx, yPD, incy);

    double endPD = omp_get_wtime();

    printf("Parallel double = %f", endPD - startPD);
    printf("\n");
    printf("\n");

    delete[] xPD;
    delete[] yPD;


		block_size = 8;
		num_blocks = (n + block_size - 1) / block_size;
		
		printf("block_size = %d\n", block_size);
		printf("num_blocks = %d\n", num_blocks);
    xF = new float[sizeX];
    yF = new float[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xF[i] = 5.0f;
        yF[i] = 1.0f;
    }
    saxpy_gpu(n, aF, xF, incx, yF, incy, num_blocks, block_size);
		printf("\n");

    delete[] xF;
    delete[] yF;

		block_size = 16;
		num_blocks = (n + block_size - 1) / block_size;
		
		printf("block_size = %d\n", block_size);
		printf("num_blocks = %d\n", num_blocks);
    xF = new float[sizeX];
    yF = new float[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xF[i] = 5.0f;
        yF[i] = 1.0f;
    }
    saxpy_gpu(n, aF, xF, incx, yF, incy, num_blocks, block_size);
		printf("\n");

    delete[] xF;
    delete[] yF;

		block_size = 32;
		num_blocks = (n + block_size - 1) / block_size;
		
		printf("block_size = %d\n", block_size);
		printf("num_blocks = %d\n", num_blocks);
    xF = new float[sizeX];
    yF = new float[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xF[i] = 5.0f;
        yF[i] = 1.0f;
    }
    saxpy_gpu(n, aF, xF, incx, yF, incy, num_blocks, block_size);
		printf("\n");

    delete[] xF;
    delete[] yF;

		block_size = 64;
		num_blocks = (n + block_size - 1) / block_size;
		
		printf("block_size = %d\n", block_size);
		printf("num_blocks = %d\n", num_blocks);
    xF = new float[sizeX];
    yF = new float[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xF[i] = 5.0f;
        yF[i] = 1.0f;
    }
    saxpy_gpu(n, aF, xF, incx, yF, incy, num_blocks, block_size);
		printf("\n");
		printf("\n");

    delete[] xF;
    delete[] yF;


		block_size = 8;
		num_blocks = (n + block_size - 1) / block_size;
		
		printf("block_size = %d\n", block_size);
		printf("num_blocks = %d\n", num_blocks);
    xD = new double[sizeX];
    yD = new double[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xD[i] = 5.0;
        yD[i] = 1.0;
    }
    daxpy_gpu(n, aD, xD, incx, yD, incy, num_blocks, block_size);
		printf("\n");

    delete[] xD;
    delete[] yD;

		block_size = 16;
		num_blocks = (n + block_size - 1) / block_size;
		
		printf("block_size = %d\n", block_size);
		printf("num_blocks = %d\n", num_blocks);
    xD = new double[sizeX];
    yD = new double[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xD[i] = 5.0;
        yD[i] = 1.0;
    }
    daxpy_gpu(n, aD, xD, incx, yD, incy, num_blocks, block_size);
		printf("\n");

    delete[] xD;
    delete[] yD;

		block_size = 32;
		num_blocks = (n + block_size - 1) / block_size;
		
		printf("block_size = %d\n", block_size);
		printf("num_blocks = %d\n", num_blocks);
    xD = new double[sizeX];
    yD = new double[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xD[i] = 5.0;
        yD[i] = 1.0;
    }
    daxpy_gpu(n, aD, xD, incx, yD, incy, num_blocks, block_size);
		printf("\n");

    delete[] xD;
    delete[] yD;

		block_size = 64;
		num_blocks = (n + block_size - 1) / block_size;
		
		printf("block_size = %d\n", block_size);
		printf("num_blocks = %d\n", num_blocks);
    xD = new double[sizeX];
    yD = new double[sizeY];

    for (int i = 0; i < n; ++i)
    {
        xD[i] = 5.0;
        yD[i] = 1.0;
    }
    daxpy_gpu(n, aD, xD, incx, yD, incy, num_blocks, block_size);
		printf("\n");

    delete[] xD;
    delete[] yD;

    return 0;
}
